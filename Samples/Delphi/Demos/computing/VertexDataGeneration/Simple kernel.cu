
#include <hip/hip_runtime.h>
////////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory'
////////////////////////////////////////////////////////////////////////////////
__global__ void kernel(
float4* pos,
unsigned int width,
unsigned int height,
float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
        float u = x / (float) width;
        float v = y / (float) height;
        u = u*2.0f - 1.0f;
        v = v*2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width+x] = make_float4(u, w, v, 1.0f);
}