#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// Texture reference for reading velocity field
texture<float2, 2> texref;

// Note that these kernels are designed to work with arbitrary
// domain sizes, not just domains that are multiples of the tile
// size. Therefore, we have extra code that checks to make sure
// a given thread location falls within the domain boundaries in
// both X and Y. Also, the domain is covered by looping over
// multiple elements in the Y direction, while there is a one-to-one
// mapping between threads in X and the tile size in X.
// Nolan Goodnight 9/22/06

// This method adds constant force vectors to the velocity field
// stored in 'v' according to v(x,t+1) = v(x,t) + dt * f.
extern "C" __global__ void
addForces_k(
float2 *v,
int dx,
int dy,
int spx,
int spy,
float fx,
float fy,
int r,
size_t pitch) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

    float2 vterm = *fj;
    tx -= r; ty -= r;
    float s = 1.f / (1.f + tx*tx*tx*tx + ty*ty*ty*ty);
    vterm.x += s * fx;
    vterm.y += s * fy;
    *fj = vterm;
}

// This method performs the velocity advection step, where we
// trace velocity vectors back in time to update each grid cell.
// That is, v(x,t+1) = v(p(x,-dt),t). Here we perform bilinear
// interpolation in the velocity space.
extern "C" __global__ void
advectVelocity_k(float *vx, float *vy,
                 int dx, int pdx, int dy, float dt, int lb) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float2 vterm, ploc;
    float vxterm, vyterm;
    // gtidx is the domain location in x for this thread
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * pdx + gtidx;
                vterm = tex2D(texref, (float)gtidx, (float)fi);
                ploc.x = (gtidx + 0.5f) - (dt * vterm.x * dx);
                ploc.y = (fi + 0.5f) - (dt * vterm.y * dy);
                vterm = tex2D(texref, ploc.x, ploc.y);
                vxterm = vterm.x; vyterm = vterm.y;
                vx[fj] = vxterm;
                vy[fj] = vyterm;
            }
        }
    }
}

// This method performs velocity diffusion and forces mass conservation
// in the frequency domain. The inputs 'vx' and 'vy' are complex-valued
// arrays holding the Fourier coefficients of the velocity field in
// X and Y. Diffusion in this space takes a simple form described as:
// v(k,t) = v(k,t) / (1 + visc * dt * k^2), where visc is the viscosity,
// and k is the wavenumber. The projection step forces the Fourier
// velocity vectors to be orthogonal to the vectors for each
// wavenumber: v(k,t) = v(k,t) - ((k dot v(k,t) * k) / k^2.
extern "C" __global__ void
diffuseProject_k(
float2 *vx,
float2 *vy,
int dx,
int dy,
float dt,
float visc,
int lb) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float2 xterm, yterm;
    // gtidx is the domain location in x for this thread
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                xterm = vx[fj];
                yterm = vy[fj];

                // Compute the index of the wavenumber based on the
                // data order produced by a standard NN FFT.
                int iix = gtidx;
                int iiy = (fi>dy/2)?(fi-(dy)):fi;

                // Velocity diffusion
                float kk = (float)(iix * iix + iiy * iiy); // k^2
                float diff = 1.f / (1.f + visc * dt * kk);
                xterm.x *= diff; xterm.y *= diff;
                yterm.x *= diff; yterm.y *= diff;

                // Velocity projection
                if (kk > 0.f) {
                    float rkk = 1.f / kk;
                    // Real portion of velocity projection
                    float rkp = (iix * xterm.x + iiy * yterm.x);
                    // Imaginary portion of velocity projection
                    float ikp = (iix * xterm.y + iiy * yterm.y);
                    xterm.x -= rkk * rkp * iix;
                    xterm.y -= rkk * ikp * iix;
                    yterm.x -= rkk * rkp * iiy;
                    yterm.y -= rkk * ikp * iiy;
                }

                vx[fj] = xterm;
                vy[fj] = yterm;
            }
        }
    }
}

// This method updates the velocity field 'v' using the two complex
// arrays from the previous step: 'vx' and 'vy'. Here we scale the
// real components by 1/(dx*dy) to account for an unnormalized FFT.
extern "C" __global__ void
updateVelocity_k(
float2 *v,
float *vx,
float *vy,
int dx,
int pdx,
int dy,
int lb,
size_t pitch,
float scale) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float vxterm, vyterm;
    float2 nvterm;
    // gtidx is the domain location in x for this thread
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
				int fjr = fi * pdx + gtidx;
                vxterm = vx[fjr];
                vyterm = vy[fjr];

                // Normalize the result of the inverse FFT
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}

// This method updates the particles by moving particle positions
// according to the velocity field and time step. That is, for each
// particle: p(t+1) = p(t) + dt * v(p(t)).
extern "C" __global__ void
advectParticles_k(
float2 *part,
float2 *v,
int dx,
int dy,
float dt,
int lb,
size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    // gtidx is the domain location in x for this thread
    float2 pterm, vterm;
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                pterm = part[fj];

                int xvi = ((int)(pterm.x * dx));
                int yvi = ((int)(pterm.y * dy));
                vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

                pterm.x += dt * vterm.x;
                pterm.x = pterm.x - (int)pterm.x;
                pterm.x += 1.f;
                pterm.x = pterm.x - (int)pterm.x;
                pterm.y += dt * vterm.y;
                pterm.y = pterm.y - (int)pterm.y;
                pterm.y += 1.f;
                pterm.y = pterm.y - (int)pterm.y;

                part[fj] = pterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}
