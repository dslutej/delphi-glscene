#include "hip/hip_runtime.h"
// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> tex;

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void
transformKernel( float* g_odata, int width, int height, float theta) 
{
    // calculate normalized texture coordinates
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    float u = x / (float) width;
    float v = y / (float) height;

    // transform coordinates
    u -= 0.5f;
    v -= 0.5f;
    float tu = u*cosf(theta) - v*sinf(theta) + 0.5f;
    float tv = v*cosf(theta) + u*sinf(theta) + 0.5f;

    // read from texture and write to global memory
    g_odata[y*width + x] = tex2D(tex, tu, tv);
}
