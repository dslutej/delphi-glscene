// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// clamp x to range [a, b]
__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

// get pixel from 2D image, with clamping to border
__device__ int getPixel(int *data, int x, int y, int width, int height)
{
    x = clamp(x, 0, width-1);
    y = clamp(y, 0, height-1);
    return data[y*width+x];
}

// macros to make indexing shared memory easier
#define SMEM(X, Y) sdata[(Y)*tilew+(X)]

/*
    2D convolution using shared memory
    - operates on 8-bit RGB data stored in 32-bit int
    - assumes kernel radius is less than or equal to block size
    - not optimized for performance
     _____________
    |   :     :   |
    |_ _:_____:_ _|
    |   |     |   |
    |   |     |   |
    |_ _|_____|_ _|
  r |   :     :   |
    |___:_____:___|
      r    bw   r
    <----tilew---->
*/

extern __shared__ int sdata[];

extern "C"
__global__ void
cudaProcess_k(
int* g_data,
int* g_odata,
int imgw,
int imgh,
int tilew,
int r,
float threshold,
float highlight)
{
    

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

    // copy tile to shared memory
    // center region
    SMEM(r + tx, r + ty) = getPixel(g_data, x, y, imgw, imgh);

    // borders
    if (threadIdx.x < r) {
        // left
        SMEM(tx, r + ty) = getPixel(g_data, x - r, y, imgw, imgh);
        // right
        SMEM(r + bw + tx, r + ty) = getPixel(g_data, x + bw, y, imgw, imgh);
    }
    if (threadIdx.y < r) {
        // top
        SMEM(r + tx, ty) = getPixel(g_data, x, y - r, imgw, imgh);
        // bottom
        SMEM(r + tx, r + bh + ty) = getPixel(g_data, x, y + bh, imgw, imgh);
    }

    // load corners
    if ((threadIdx.x < r) && (threadIdx.y < r)) {
        // tl
        SMEM(tx, ty) = getPixel(g_data, x - r, y - r, imgw, imgh);
        // bl
        SMEM(tx, r + bh + ty) = getPixel(g_data, x - r, y + bh, imgw, imgh);
        // tr
        SMEM(r + bw + tx, ty) = getPixel(g_data, x + bh, y - r, imgw, imgh);
        // br
        SMEM(r + bw + tx, r + bh + ty) = getPixel(g_data, x + bw, y + bh, imgw, imgh);
    }

    // wait for loads to complete
    __syncthreads();

    // perform convolution
    float rsum = 0.0;
    float gsum = 0.0;
    float bsum = 0.0;
    float samples = 0.0;

    for(int dy=-r; dy<=r; dy++) {
        for(int dx=-r; dx<=r; dx++) {
			int pixel = SMEM(r+tx+dx, r+ty+dy);

            // only sum pixels within disc-shaped kernel
            float l = dx*dx + dy*dy;
			if (l <= r*r)
			{
                float r = float(pixel&0xff);
                float g = float((pixel>>8)&0xff);
                float b = float((pixel>>16)&0xff);

//                // brighten highlights
//                float lum = (r + g + b) / (255*3);
//				if (lum > threshold)
//				{
//                    r *= highlight;
//					g *= highlight;
//                    b *= highlight;
//				}

                rsum += r;
                gsum += g;
                bsum += b;
                samples += 1.0;
            }
        }
    }

    rsum /= samples;
    gsum /= samples;
    bsum /= samples;
    g_odata[y*imgw+x] = rgbToInt(rsum, gsum, bsum);
}
